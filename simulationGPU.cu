#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "simulationGPU.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>

using namespace std;

// Initialisation and destruction of class and it's attributes and methods: -------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

simulationGPU::simulationGPU(float densityInp, int numXInp, int numYInp, float hInp) {
    density = densityInp;
    numX = numXInp + 2;
    numY = numYInp + 2;
    numCells = numX * numY;
    h = hInp;
    numRows = numY;
    inletVelocity = 2.0f;
    relativeInletHeight = 0.14f;
    shape = 0;

    // Allocate device memory
    hipMalloc(&d_u,     numCells * sizeof(float));
    hipMalloc(&d_v,     numCells * sizeof(float));
    hipMalloc(&d_newU,  numCells * sizeof(float));
    hipMalloc(&d_newV,  numCells * sizeof(float));
    hipMalloc(&d_m,     numCells * sizeof(float));
    hipMalloc(&d_newM,  numCells * sizeof(float));
    hipMalloc(&d_p,     numCells * sizeof(float));
    hipMalloc(&d_s,     numCells * sizeof(float));

    // Initialize with zeros
    std::vector<float> temp0(numCells, 0.0f);
    std::vector<float> temp1(numCells, 1.0f);

    hipMemcpy(d_u, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, temp1.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_p, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_newU, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_newV, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_newM, temp0.data(), numCells * sizeof(float), hipMemcpyHostToDevice);
}

simulationGPU::~simulationGPU() {
    hipFree(d_u); // velocity in the horizontal direction
    hipFree(d_v); // velocity in the vertical direction
    hipFree(d_newU);
    hipFree(d_newV);
    hipFree(d_m); // density of each cell in the grid
    hipFree(d_newM);
    hipFree(d_p); // pressure of each cell i the grid
    hipFree(d_s); // whether each cell is fluid or solid
}

// Helper functions, all used only in GPU: ----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

__device__ inline int gridIndex(int i, int j, int numY) {return ((i * numY) + j);}

__device__ float avgU(int i, int j, float *u, int numY) {
    return (u[gridIndex(i,j - 1,numY)] + u[gridIndex(i,j,numY)] + u[gridIndex(i + 1, j - 1,numY)] + u[gridIndex(i + 1,j,numY)]) * 0.25;
}

__device__ float avgV(int i, int j, float *v, int numY) {
    return (v[gridIndex(i,j - 1,numY)] + v[gridIndex(i,j,numY)] + v[gridIndex(i + 1, j - 1,numY)] + v[gridIndex(i + 1,j,numY)]) * 0.25;
}

__device__ float sampleField(float x, float y, int fieldType, float *u, float *v, float *m, float h, int numX, int numY) {
    float h1 = 1.0 / h;
    float h2 = 0.5 * h;

    // Clamp world coordinates to valid range
    x = max(min(x, numX * h), h);
    y = max(min(y, numY * h), h);

    float dx = 0.0;
    float dy = 0.0;
    const float *f = nullptr;

    switch (fieldType) {
        case 0: f = u; dy = h2; break;
        case 1: f = v; dx = h2; break;
        case 2: f = m; dx = h2; dy = h2; break;
    }

    // Convert world coords -> grid indices
    float x0 = min(floor((x - dx) * h1), numX - 1.0);
    float tx = ((x - dx) - x0 * h) * h1;
    float x1 = min(x0 + 1, numX - 1.0);

    float y0 = min(floor((y - dy) * h1), numY - 1.0);
    float ty = ((y - dy) - y0 * h) * h1;
    float y1 = min(y0 + 1, numY - 1.0);

    float sx = 1.0f - tx;
    float sy = 1.0f - ty;

    // Bilinear interpolation using 1D indexing
    return (sx * sy * f[gridIndex(x0,y0,numY)]) +
           (tx * sy * f[gridIndex(x1,y0,numY)]) +
           (tx * ty * f[gridIndex(x1,y1,numY)]) +
           (sx * ty * f[gridIndex(x0,y1,numY)]);
}

__device__ float nacaThickness(float x, float chord, float t) {
    float X = x / chord;
    return 0.5f * t * chord * (0.2969f*sqrtf(X) - 0.1260f*X - 0.3516f*X*X + 0.2843f*X*X*X - 0.1015f*X*X*X*X);
}

// GPU kernels: -------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

    // Kernels for the fluid simulations:

__global__ void integrate(float *v, float *s, int numX, int numY, float gravity, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= 1 && i < numX && j >= 1 && j < numY - 1) {
        if (s[gridIndex(i,j,numY)] != 0.0f && s[gridIndex(i,j - 1,numY)] != 0.0f) {
            v[gridIndex(i,j,numY)] += gravity * dt;
        }
    }
}

__global__ void clearOldPressures(int numX, int numY, float *p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= numX || j >= numY) return;
    p[gridIndex(i,j,numY)] = 0.0f;
}

__global__ void solveIncompressibility(int grid, float cp, float *s, float *u, float *v, float *p,int numX, int numY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i + j) % 2 != grid) {return;}

    if (i > 0 && i < numX - 1 && j > 0 && j < numY - 1) {
        if (s[gridIndex(i,j,numY)] == 0.0f) {return;}

        float sTemp = s[gridIndex(i,j,numY)];
        float sx0 = s[gridIndex(i - 1,j,numY)];
        float sx1 = s[gridIndex(i + 1,j,numY)];
        float sy0 = s[gridIndex(i,j - 1,numY)];
        float sy1 = s[gridIndex(i,j + 1,numY)];
        sTemp = sx0 + sx1 + sy0 + sy1;
        if (sTemp == 0.0f) {return;}

        float divergence = u[gridIndex(i + 1,j,numY)] - u[gridIndex(i,j,numY)] + v[gridIndex(i,j + 1,numY)] - v[gridIndex(i,j,numY)];
        float dTemp = -divergence / sTemp;
        dTemp *= 1.9;

        p[gridIndex(i,j,numY)] += cp * dTemp;
        u[gridIndex(i,j,numY)] -= sx0 * dTemp;
        u[gridIndex(i + 1,j,numY)] += sx1 * dTemp;
        v[gridIndex(i,j,numY)] -= sy0 * dTemp;
        v[gridIndex(i,j + 1,numY)] += sy1 * dTemp;
    }
}

__global__ void extrapolateU(int numX, int numY, float *u) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numX) return;

    u[gridIndex(i,0,numY)] = u[gridIndex(i,1,numY)];
    u[gridIndex(i,numY - 1,numY)] = u[gridIndex(i,numY - 2,numY)];
}

__global__ void extrapolateV(int numX, int numY, float *v) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numY) return;

    v[gridIndex(0,i,numY)] = v[gridIndex(1,i,numY)];
    v[gridIndex(numX - 1,i,numY)] = v[gridIndex(numX - 2,i,numY)];
}

__global__ void advectVel(float dt, float h, int numX, int numY, float *u, float *v, float *s, float *m, float *newU, float *newV) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 0 || i >= numX - 1 || j <= 0 || j >= numY - 1) return; // is the thread in the required grid?

    float h2 = 0.5 * h;

    if (s[gridIndex(i,j,numY)] != 0.0f && s[gridIndex(i - 1,j,numY)] != 0.0f) {
        float x = i * h;
        float y = (j * h) + h2;
        float uTemp = u[gridIndex(i,j,numY)];
        float vTemp = avgV(i,j,v,numY);

        x-= (dt * uTemp);
        y-= (dt * vTemp);
        uTemp = sampleField(x,y,0,u,v,m,h,numX,numY);

        newU[gridIndex(i,j,numY)] = uTemp;
    }
    if (s[gridIndex(i,j,numY)] != 0.0f && s[gridIndex(i,j - 1,numY)] != 0.0f) {
        float x = (i * h) + h2;
        float y = j * h;
        float uTemp = avgU(i,j,u,numY);
        float vTemp = v[gridIndex(i,j,numY)];

        x-= (dt * uTemp);
        y-= (dt * vTemp);
        vTemp = sampleField(x,y,1,u,v,m,h,numX,numY);

        newV[gridIndex(i,j,numY)] = vTemp;
    }
}

__global__ void advectSmoke(float dt, float h, float *u, float *v, float *s, float *m, float *newM, int numX, int numY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 0 || i >= numX - 1 || j <= 0 || j >= numY - 1) return;

    float h2 = 0.5 * h;

    if (s[gridIndex(i,j,numY)] != 0.0f) {
        float uTemp = (u[gridIndex(i,j,numY)] + u[gridIndex(i + 1,j,numY)]) * 0.5;
        float vTemp = (v[gridIndex(i,j,numY)] + v[gridIndex(i,j + 1,numY)]) * 0.5;
        float x = (i * h) + h2 - (dt * uTemp);
        float y = (j * h) + h2 - (dt * vTemp);

        newM[gridIndex(i,j,numY)] = sampleField(x,y,2,u,v,m,h,numX,numY);
    }
}

    // Kernels for setting up the environment in the application:

__global__ void setUpSceneMemory(int numX, int numY, float *s, float *u, float *v, float *m, float inletVelocity, float relativeInletHeight) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= numX || j >= numY) return;

    int idx = gridIndex(i,j,numY);

    float initialVelocity = inletVelocity;

    float sTemp = 1.0f; // Fluid
    if (i == 0 || j == 0 || j == numY -1) {
        sTemp = 0.0; // Solid
    }

    s[idx] = sTemp;

    if (i == 1) {
        u[idx] = initialVelocity;
    }

    // Inlet region marker
    float inletHeight = relativeInletHeight * numY;
    int minHeight = floor((0.5 * numY) - (0.5 * inletHeight));
    int maxHeight = floor((0.5 * numY) + (0.5 * inletHeight));
    if (j >= minHeight && j < maxHeight && i == 0) {
        m[idx] = 0.0f; // mark as smoky for inlet
    }
    if ((j < minHeight || j >= maxHeight) && i == 0) {
        m[idx] = 1.0f;
    }

}

__global__ void setUpCircleObstacle(int numX, int numY, float *s, float *u, float *v, float *m, float xNorm, float yNorm, float h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 0 || i >= numX - 1 || j <= 0 || j >= numY - 1) return;

    float worldX = xNorm * (numX - 2) * h;
    float worldY = yNorm * (numY - 2) * h;
    float radius = 0.1f * (numX - 2) * h;
    float r2 = radius * radius;

    float cellX = (i + 0.5f) * h;
    float cellY = (j + 0.5f) * h;
    float dx = cellX - worldX;
    float dy = cellY - worldY;

    if ((dx*dx + dy*dy) < (r2)) {
        int idx = gridIndex(i,j,numY);
        s[idx] = 0.0f;   // mark solid
        m[idx] = 1.0f;   // smoke/marker solid

        u[idx] = u[gridIndex(i + 1, j, numY)] = 0.0f;
        v[idx] = v[gridIndex(i, j + 1, numY)] = 0.0f;
    }
}

__global__ void setUpEllipseObstacle(int numX, int numY, float *s, float *u, float *v, float *m, float xNorm, float yNorm, float h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 0 || i >= numX - 1 || j <= 0 || j >= numY - 1) return;

    float worldX = xNorm * (numX - 2) * h;
    float worldY = yNorm * (numY - 2) * h;
    float radiusX = 0.16f * (numX - 2) * h;
    float radiusY = 0.12f * (numY - 2) * h;

    float cellX = (i + 0.5f) * h;
    float cellY = (j + 0.5f) * h;
    float dx = (cellX - worldX) / radiusX;
    float dy = (cellY - worldY) / radiusY;


    if (dx*dx + dy*dy < 1.0f) {
        int idx = gridIndex(i,j,numY);
        s[idx] = 0.0f;   // mark solid
        m[idx] = 1.0f;   // smoke/marker solid

        u[idx] = u[gridIndex(i + 1, j, numY)] = 0.0f;
        v[idx] = v[gridIndex(i, j + 1, numY)] = 0.0f;
    }
}

__global__ void setUpSquareObstacle(int numX, int numY, float *s, float *u, float *v, float *m, float xNorm, float yNorm, float h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 0 || i >= numX - 1 || j <= 0 || j >= numY - 1) return;

    // Center of the square in world coordinates
    float worldX = xNorm * (numX - 2) * h;
    float worldY = yNorm * (numY - 2) * h;

    // Half side length in world coordinates
    float halfSide = 0.5f * 0.16f * (numX - 2) * h;

    float cellX = (i + 0.5f) * h;
    float cellY = (j + 0.5f) * h;
    float dx = fabsf(cellX - worldX);
    float dy = fabsf(cellY - worldY);

    // Square test
    if (dx <= halfSide && dy <= halfSide) {
        int idx = gridIndex(i, j, numY);
        s[idx] = 0.0f;   // mark solid
        m[idx] = 1.0f;   // smoke/marker solid

        u[idx] = u[gridIndex(i + 1, j, numY)] = 0.0f;
        v[idx] = v[gridIndex(i, j + 1, numY)] = 0.0f;
    }
}

__global__ void setUpWingObstacle(int numX, int numY, float *s, float *u, float *v, float *m, float xNorm, float yNorm, float h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 0 || i >= numX - 1 || j <= 0 || j >= numY - 1) return;

    float worldX = xNorm * (numX - 2) * h; // leading edge
    float worldY = yNorm * (numY - 2) * h; // flat bottom reference

    float chord = 0.3f * (numX - 2) * h;      // chord length
    float thickness = 0.5f * (numY - 2) * h; // thick
    float angleDeg = 6.0f;                   // pitch angle
    float angleRad = angleDeg * 3.14159265f / 180.0f;

    // Flip y-axis so j=0 is bottom
    float cellY = (numY - 1 - j + 0.5f) * h;
    float cellX = (i + 0.5f) * h;
    float xLocal = cellX - worldX;

    if (xLocal >= 0.0f && xLocal <= chord) {
        float yBottom = worldY + xLocal * -tanf(angleRad); // tilt the bottom
        float yTop = yBottom + nacaThickness(xLocal, chord, thickness); // curved top

        if (cellY >= yBottom && cellY <= yTop) {
            int idx = gridIndex(i, j, numY);
            s[idx] = 0.0f;   // mark solid
            m[idx] = 1.0f;   // marker solid

            u[idx] = u[gridIndex(i + 1, j, numY)] = 0.0f;
            v[idx] = v[gridIndex(i, j + 1, numY)] = 0.0f;
        }
    }
}

// Runner methods: ----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void simulationGPU::runIntegration(float dt, float gravity) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((numX + 15) / 16, (numY + 15) / 16);

    integrate<<<numBlocks, threadsPerBlock>>>(d_v, d_s, numX, numY, gravity, dt);
    hipDeviceSynchronize();
}

void simulationGPU::runClearOldPressures() {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((numX + 15) / 16, (numY + 15) / 16);

    clearOldPressures<<<numBlocks, threadsPerBlock>>>(numX, numY,d_p);
    hipDeviceSynchronize();
}

void simulationGPU::runSolveIncompressibility(int numIterations, float dt) {
    float cp = density * h / dt;
    dim3 blockSize(16,16);
    dim3 gridSize((numX + 15)/16, (numY + 15)/16);
    for (int iter = 0; iter < numIterations; iter++) {
        solveIncompressibility<<<gridSize, blockSize>>>(0, cp, d_s, d_u, d_v, d_p, numX, numY);
        hipDeviceSynchronize();

        solveIncompressibility<<<gridSize, blockSize>>>(1, cp, d_s, d_u, d_v, d_p, numX, numY);
        hipDeviceSynchronize();
    }
}

void simulationGPU::runExtrapolation() {
    dim3 blockSize(128); // 1D thread block
    dim3 numBlocksU((numX + blockSize.x - 1) / blockSize.x);
    dim3 numBlocksV((numY + blockSize.x - 1) / blockSize.x);

    extrapolateU<<<numBlocksU, blockSize>>>(numX, numY, d_u);
    extrapolateV<<<numBlocksV, blockSize>>>(numX, numY, d_v);
    hipDeviceSynchronize();
}

void simulationGPU::runAdvections(float dt) {
    dim3 blockSize(16,16);
    dim3 gridSize((numX + 15)/16, (numY + 15)/16);

    // --- Copy current fields into "new" buffers at the beginning ---
    hipMemcpy(d_newU, d_u, numX * numY * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_newV, d_v, numX * numY * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_newM, d_m, numX * numY * sizeof(float), hipMemcpyDeviceToDevice);

    // --- Advection kernels ---
    advectVel<<<gridSize, blockSize>>>(dt, h, numX, numY, d_u, d_v, d_s, d_m, d_newU, d_newV);
    hipDeviceSynchronize();

    advectSmoke<<<gridSize, blockSize>>>(dt, h, d_u, d_v, d_s, d_m, d_newM, numX, numY);
    hipDeviceSynchronize();

    std::swap(d_u, d_newU);
    std::swap(d_v, d_newV);
    std::swap(d_m, d_newM);

}


// Getter methods, for UI grids: --------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void simulationGPU::getVelocityGrids(vector<float>& u, vector<float>& v) {
    u.resize(numCells);
    v.resize(numCells);

    hipMemcpy(u.data(), d_u, numCells * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(v.data(), d_v, numCells * sizeof(float), hipMemcpyDeviceToHost);
}

void simulationGPU::getSmokeDensityGrid(vector<float>& m) {
    m.resize(numCells);
    hipMemcpy(m.data(), d_m, numCells * sizeof(float), hipMemcpyDeviceToHost);
}

void simulationGPU::getPressureGrid(vector<float>& p) {
    p.resize(numCells);
    hipMemcpy(p.data(), d_p, numCells * sizeof(float), hipMemcpyDeviceToHost);
}

void simulationGPU::getSolidFluidGrid(vector<float>& s) {
    s.resize(numCells);
    hipMemcpy(s.data(), d_s, numCells * sizeof(float), hipMemcpyDeviceToHost);
}

// Setting up and changing environment / scene: ------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void simulationGPU::setScene() {
    dim3 blockSize(16,16);
    dim3 gridSize((numX + 15)/16, (numY + 15)/16);
    // Initialize the scene
    setUpSceneMemory<<<gridSize, blockSize>>>(numX, numY, d_s, d_u, d_v, d_m, inletVelocity, relativeInletHeight);
    hipDeviceSynchronize();

    // Add obstacle at normalized coordinates (0.3, 0.5)
    switch (shape) {
        case 0:
            setUpCircleObstacle<<<gridSize, blockSize>>>(numX, numY, d_s, d_u, d_v, d_m, 0.3f, 0.5f, h);
            break;
        case 1:
            setUpEllipseObstacle<<<gridSize, blockSize>>>(numX, numY, d_s, d_u, d_v, d_m, 0.3f, 0.5f, h);
            break;
        case 2:
            setUpSquareObstacle<<<gridSize, blockSize>>>(numX, numY, d_s, d_u, d_v, d_m, 0.3f, 0.5f, h);
            break;
        case 3:
            setUpWingObstacle<<<gridSize, blockSize>>>(numX, numY, d_s, d_u, d_v, d_m, 0.3f, 0.5f, h);
            break;
    }
    hipDeviceSynchronize();
}

void simulationGPU::updateShape(int shapeInp) {
    shape = shapeInp;
    setScene();
}

void simulationGPU::updateInletVel(float inletVelocityInp) {
    inletVelocity = inletVelocityInp;
    setScene();
}

void simulationGPU::updateInletSize(float inletSizeInp) {
    relativeInletHeight = inletSizeInp;
    setScene();
}

// Simulation coordination, singular run: -----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void simulationGPU::simulate(float dt, float gravity, int numIterations) {
    //runIntegration(dt, gravity);  don't need gravity for wind tunnel simulations

    runClearOldPressures();

    runSolveIncompressibility(numIterations, dt);

    runExtrapolation();

    runAdvections(dt);

    hipDeviceSynchronize(); // wait for GPU
}

// Credits:
// Portions of this code were inspired by:
// Copyright 2022 Matthias Müller - Ten Minute Physics
// MIT License
// Website: https://www.matthiasMueller.info/tenMinutePhysics
// YouTube: https://www.youtube.com/c/TenMinutePhysics
